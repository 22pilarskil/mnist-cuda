#include "../../include/model.h"
#include "../../include/macros.h"
#include "../../include/utils.h"
#include "../../include/layers/leaky_relu.h"
#include <stdio.h>


Layer* initLeakyReLU(int batch_size, int dim, int coeff, float* inputs) {
    Layer* layer = (Layer*)malloc(sizeof(Layer));
    LeakyReLU* leakyReLU = (LeakyReLU*)malloc(sizeof(LeakyReLU));
    leakyReLU->coeff = coeff;
    leakyReLU->dim = dim;

    hipMallocManaged(&layer->outputs, batch_size * dim * sizeof(float));

    layer->forward = leakyReLU_forward;
    layer->backward = leakyReLU_backward;
    layer->update = leakyReLU_update;
    layer->weights_size = 0;
    MALLOC(&layer->downstream_grads, batch_size * dim * sizeof(float));
    layer->inputs = inputs;
    layer->layer_data = leakyReLU;
    layer->type = LAYER_LEAKY_RELU;
    return layer;
}


void leakyReLU_forward(Layer* layer, int batch_size) { 
    LeakyReLU* leakyReLU = (LeakyReLU*)layer->layer_data;       
    host_leakyReLU_forward(layer->inputs, layer->outputs, batch_size, leakyReLU->dim, leakyReLU->coeff);

}


void host_leakyReLU_forward(float* inputs, float* outs, int batch_size, int dim, float coeff) {
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < dim; j++) {
            int idx = i * dim + j;
            outs[idx] = (inputs[idx] > 0) ? inputs[idx] : (coeff * inputs[idx]);
        }
    }
}

void leakyReLU_backward(Layer* layer, int batch_size) {
    LeakyReLU* leakyReLU = (LeakyReLU*)layer->layer_data;
    host_leakyReLU_backward(layer, leakyReLU, batch_size);
}

void host_leakyReLU_backward(Layer* layer, LeakyReLU* leakyReLU, int batch_size) {
    int dim = leakyReLU->dim;
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < dim; j++) {
            int idx = i * dim + j;
            if (layer->inputs[idx] >= 0) {
                layer->downstream_grads[idx] = layer->upstream_grads[idx];
            } else {
                layer->downstream_grads[idx] = leakyReLU->coeff * layer->upstream_grads[idx];
            }
        }
    }
}

void leakyReLU_update(Layer* layer, int batch_size) {
    
}